#include "hip/hip_runtime.h"
#include <iostream>
#include <sgemm.cuh>
#include <mat_tools.cuh>

#define cudaCheck(err) (cudaCheck(err, __FILE__, __LINE__))

int div_ceil(int numerator, int denominator) {
  std::div_t res = std::div(numerator, denominator);
  return res.rem ? (res.quot + 1) : res.quot;
}


__global__ void sgemm_naive(int M, int N, int K, float alpha,
                            float *A, float *B, float beta, float *C)
{
  // printf("Kernel running on block : %u thread : %u\n", blockIdx.x, threadIdx.x);
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  if(x < N && y < M)
  {
    float tmp = 0.0f;
    int i;
    for (i = 0; i < K; i++)
      tmp += A[y * K + i] * B[i * N + x];
    C[y * N + x] = alpha * tmp + beta * C[y * N + x];
  }
}

void sgemm_gpu(int M, int N, int K, float alpha,
               float *A, float *B, float beta, float *C)
{
  float *d_A, *d_B, *d_C;
  dim3 gridDim(div_ceil(N, 32), div_ceil(M, 32), 1);
  dim3 blockDim(BDIMX, BDIMY, 1);
  
  cudaCheck(hipMalloc(&d_A, M * K * sizeof(float)));
  cudaCheck(hipMalloc(&d_B, N * K * sizeof(float)));
  cudaCheck(hipMalloc(&d_C, N * M * sizeof(float)));

  cudaCheck(hipMemcpy(d_A, A, M * K * sizeof(float), hipMemcpyHostToDevice));
  cudaCheck(hipMemcpy(d_B, B, N * K * sizeof(float), hipMemcpyHostToDevice));
  cudaCheck(hipMemcpy(d_C, C, M * N * sizeof(float), hipMemcpyHostToDevice));

  sgemm_naive<<<gridDim, blockDim>>>(M, N, K, alpha, d_A, d_B, beta, d_C);

  cudaCheck(hipMemcpy(C, d_C, M * N * sizeof(float), hipMemcpyDeviceToHost));

  cudaCheck(hipFree(d_A));
  cudaCheck(hipFree(d_B));
  cudaCheck(hipFree(d_C));
}

void sgemm_cpu(int M, int N, int K, float alpha,
               float *A, float *B, float beta, float *C)
{
  int x, y, i;
  float tmp;
  for (y = 0; y < M; y++)
    {
      for (x = 0; x < N; x++)
      {
        tmp = 0.0f;
        for (i = 0; i < K; i++)
          tmp += A[y * K + i] * B[i * N + x];
        C[y * N + x] = alpha * tmp + beta * C[y * N + x];
      }
    }
}