#include "hip/hip_runtime.h"
#include <iostream>
#include <sgemm.cuh>
#include <mat_tools.cuh>
#include <chrono>

#define BLOCKSIZE 32

#define cudaCheck(err) (cudaCheck(err, __FILE__, __LINE__))

int div_ceil(int numerator, int denominator) {
  std::div_t res = std::div(numerator, denominator);
  return res.rem ? (res.quot + 1) : res.quot;
}


__global__ void sgemm_naive_kernel(int M, int N, int K, float alpha,
                            float *A, float *B, float beta, float *C)
{
  // printf("Kernel running on block : %u thread : %u\n", blockIdx.x, threadIdx.x);
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  if(x < N && y < M)
  {
    float tmp = 0.0f;
    int i;
    for (i = 0; i < K; i++)
      tmp += A[y * K + i] * B[i * N + x];
    C[y * N + x] = alpha * tmp + beta * C[y * N + x];
  }
}

__global__ void siboehm_naive_kernel(int M, int N, int K, float alpha,
                              float *A, float *B, float beta, float *C)
{
  // compute position in C that this thread is responsible for
  const uint x = blockIdx.x * blockDim.x + threadIdx.x;
  const uint y = blockIdx.y * blockDim.y + threadIdx.y;

  // `if` condition is necessary for when M or N aren't multiples of 32.
  if (x < M && y < N)
  {
    float tmp = 0.0;
    for (int i = 0; i < K; ++i)
    {
      tmp += A[x * K + i] * B[i * N + y];
    }
    // C = α*(A@B)+β*C
    C[x * N + y] = alpha * tmp + beta * C[x * N + y];
  }
}

__global__ void sgemm_coalesced_kernel(int M, int N, int K, float alpha,
                                float *A, float *B, float beta, float *C)
{
  // compute position in C that this thread is responsible for
  const int x = blockIdx.x * BLOCKSIZE + (threadIdx.x / BLOCKSIZE);
  const int y = blockIdx.y * BLOCKSIZE + (threadIdx.x % BLOCKSIZE);

  // `if` condition is necessary for when M or N aren't multiples of 32.
  if (x < M && y < N)
  {
    float tmp = 0.0;
    for (int i = 0; i < K; ++i)
    {
      tmp += A[x * K + i] * B[i * N + y];
    }
    // C = α*(A@B)+β*C
    C[x * N + y] = alpha * tmp + beta * C[x * N + y];
  }  
}

void sgemm_gpu_naive(int M, int N, int K, float alpha,
               float *A, float *B, float beta, float *C)
{
  float *d_A, *d_B, *d_C;
  dim3 gridDim(div_ceil(N, 32), div_ceil(M, 32), 1);
  dim3 blockDim(32, 32, 1);
  
  cudaCheck(hipMalloc(&d_A, M * K * sizeof(float)));
  cudaCheck(hipMalloc(&d_B, N * K * sizeof(float)));
  cudaCheck(hipMalloc(&d_C, N * M * sizeof(float)));

  cudaCheck(hipMemcpy(d_A, A, M * K * sizeof(float), hipMemcpyHostToDevice));
  cudaCheck(hipMemcpy(d_B, B, N * K * sizeof(float), hipMemcpyHostToDevice));
  cudaCheck(hipMemcpy(d_C, C, M * N * sizeof(float), hipMemcpyHostToDevice));
    
  siboehm_naive_kernel<<<gridDim, blockDim>>>(M, N, K, alpha, d_A, d_B, beta, d_C);

  cudaCheck(hipMemcpy(C, d_C, M * N * sizeof(float), hipMemcpyDeviceToHost));

  cudaCheck(hipFree(d_A));
  cudaCheck(hipFree(d_B));
  cudaCheck(hipFree(d_C));
}

void sgemm_gpu_coal(int M, int N, int K, float alpha,
float *A, float *B, float beta, float *C)
{
  float *d_A, *d_B, *d_C;
  dim3 gridDim(div_ceil(N, 32), div_ceil(M, 32), 1);
  dim3 blockDim(BLOCKSIZE * BLOCKSIZE);
  
  cudaCheck(hipMalloc(&d_A, M * K * sizeof(float)));
  cudaCheck(hipMalloc(&d_B, N * K * sizeof(float)));
  cudaCheck(hipMalloc(&d_C, N * M * sizeof(float)));

  cudaCheck(hipMemcpy(d_A, A, M * K * sizeof(float), hipMemcpyHostToDevice));
  cudaCheck(hipMemcpy(d_B, B, N * K * sizeof(float), hipMemcpyHostToDevice));
  cudaCheck(hipMemcpy(d_C, C, M * N * sizeof(float), hipMemcpyHostToDevice));
    
  sgemm_coalesced_kernel<<<gridDim, blockDim>>>(M, N, K, alpha, d_A, d_B, beta, d_C);

  cudaCheck(hipMemcpy(C, d_C, M * N * sizeof(float), hipMemcpyDeviceToHost));

  cudaCheck(hipFree(d_A));
  cudaCheck(hipFree(d_B));
  cudaCheck(hipFree(d_C));
}


void sgemm_cpu(int M, int N, int K, float alpha,
               float *A, float *B, float beta, float *C)
{
  int x, y, i;
  float tmp;
  for (y = 0; y < M; y++)
    {
      for (x = 0; x < N; x++)
      {
        tmp = 0.0f;
        for (i = 0; i < K; i++)
          tmp += A[y * K + i] * B[i * N + x];
        C[y * N + x] = alpha * tmp + beta * C[y * N + x];
      }
    }
}